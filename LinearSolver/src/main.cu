﻿#include "hip/hip_runtime.h"
#include ""
#include "Typedef.h"
#include "LinearSolver.h"
#include "tools/BenchMark.h"

#include "CG/Parallel/P_CG.h"
#include "CG/Sequence/S_CG.h"
#include "BiCGSTAB/Parallel/P_BiCGSTAB.h"
#include "BiCGSTAB/Sequence/S_BICGSTAB.h"

double sequence[5000];
double parallel[5000];

int main()
{
	CSR data;
	data.m_Dimension = 5000;
	data.m_Data = new double[3 * data.m_Dimension - 2];
	data.m_ArrayLength = 3 * data.m_Dimension - 2;
	data.m_Data[0] = 1.0;
	for (uint32 i = 0; i < 3 * data.m_Dimension - 2; ++i)
	{
		if (i % 3 == 1)
			data.m_Data[i] = (int)(i / 3) + 2;
		else if (i % 3 == 2)
			data.m_Data[i] = (int)(i / 3) + 2;
		else
			data.m_Data[i] = (int)(i / 3) + 1;
	}

	data.m_Col = new uint32[3 * data.m_Dimension - 2];
	data.m_Col[0] = 0;
	data.m_Col[1] = 1;
	for (uint32 i = 0; i < 3 * data.m_Dimension - 4; ++i)
	{
		if (i % 3 == 2)
			data.m_Col[i] = (int)(i / 3);
		else if (i % 3 == 0)
			data.m_Col[i] = (int)(i / 3);
		else
			data.m_Col[i] = (int)(i / 3) + 1;
	}
	data.m_Col[3 * data.m_Dimension - 4] = data.m_Dimension - 2;
	data.m_Col[3 * data.m_Dimension - 3] = data.m_Dimension - 1;

	data.m_Fnz = new uint32[data.m_Dimension + 1];
	data.m_Fnz[0] = 0;
	data.m_Fnz[1] = 2;
	for (uint32 i = 2; i < data.m_Dimension; ++i)
	{
		data.m_Fnz[i] = data.m_Fnz[i - 1] + 3;
	}
	data.m_Fnz[data.m_Dimension] = 3 * data.m_Dimension - 2;

	double* x = new double[data.m_Dimension];
	for (uint32 i = 0; i < data.m_Dimension; ++i)
	{
		x[i] = 1.0 / (i + 3.14);
	}

	double* b = new double[data.m_Dimension];
	for (uint32 i = 0; i < data.m_Dimension; ++i)             // 初始化b为0
		b[i] = 0;

	Matrix_multi_Vector(data, x, b);

	for (uint32 i = 0; i < data.m_Dimension; ++i)
	{
		x[i] = 0;
	}

	uint32 count = 100000000;
	double* vector1 = new double[count];

	for (uint32 i = 0; i < count; ++i)
	{
		vector1[i] = 1;
	}

	LinearSolver ls(data, vector1, vector1, count, 50000, 0.000001);
	ls.test();

	P_LinearSolver p_ls(data, vector1, vector1, count, 50000, 0.000001);
	p_ls.test();


	uint32 error = 0;
	for (uint32 i = 0; i < 5000; ++i)
	{
		if (parallel[i] != sequence[i])
		{
			std::cout << parallel[i] << "   " << sequence[i] << std::endl;
			++error;
		}
	}
	std::cout << "error: " << error << std::endl;


	delete[] x;
	delete[] b;

	return 0;

}

int test1()
{
	CSR data;
	data.m_Dimension = 5000;
	data.m_Data = new double[3 * data.m_Dimension - 2];
	data.m_ArrayLength = 3 * data.m_Dimension - 2;
	data.m_Data[0] = 1.0;
	for (uint32 i = 0; i < 3 * data.m_Dimension - 2; ++i)
	{
		if (i % 3 == 1)
			data.m_Data[i] = (int)(i / 3) + 2;
		else if (i % 3 == 2)
			data.m_Data[i] = (int)(i / 3) + 2;
		else
			data.m_Data[i] = (int)(i / 3) + 1;
	}

	data.m_Col = new uint32[3 * data.m_Dimension - 2];
	data.m_Col[0] = 0;
	data.m_Col[1] = 1;
	for (uint32 i = 0; i < 3 * data.m_Dimension - 4; ++i)
	{
		if (i % 3 == 2)
			data.m_Col[i] = (int)(i / 3);
		else if (i % 3 == 0)
			data.m_Col[i] = (int)(i / 3);
		else
			data.m_Col[i] = (int)(i / 3) + 1;
	}
	data.m_Col[3 * data.m_Dimension - 4] = data.m_Dimension - 2;
	data.m_Col[3 * data.m_Dimension - 3] = data.m_Dimension - 1;

	data.m_Fnz = new uint32[data.m_Dimension + 1];
	data.m_Fnz[0] = 0;
	data.m_Fnz[1] = 2;
	for (uint32 i = 2; i < data.m_Dimension; ++i)
	{
		data.m_Fnz[i] = data.m_Fnz[i - 1] + 3;
	}
	data.m_Fnz[data.m_Dimension] = 3 * data.m_Dimension - 2;

	double* x = new double[data.m_Dimension];
	for (uint32 i = 0; i < data.m_Dimension; ++i)
	{
		x[i] = 1.0 / (i + 3.14);
	}

	double* b = new double[data.m_Dimension];
	for (uint32 i = 0; i < data.m_Dimension; ++i)             // 初始化b为0
		b[i] = 0;

	Matrix_multi_Vector(data, x, b);

	for (uint32 i = 0; i < data.m_Dimension; ++i)
	{
		x[i] = 0;
	}


	//double* vector1 = new double[data.m_Dimension];

	LinearSolver ls(std::move(data), x, b, 5000, 50000, 0.000001);
	ls.test();



	//P_CG solverP_CG(std::move(data), x, b, 5000, 50000, 0.000001);
	//solverP_CG.start();
	//std::cout << solverP_CG.getTime() << std::endl;
	//std::cout << solverP_CG.getIter() << std::endl;

	//S_CG solverS_CG(std::move(data), x, b, 5000, 50000, 0.000001);
	//solverS_CG.start();
	//std::cout << solverS_CG.getTime() << std::endl;
	//std::cout << solverS_CG.getIter() << std::endl;

	//CSR data2("../res/A.txt");

	//uint32 dimension = data2.m_Dimension;

	//for (uint32 i = 0; i < data2.m_ArrayLength; ++i)
	//{
	//	std::cout << data2.m_Data[i] << " ";
	//}
	//std::cout << std::endl;

	//for (uint32 i = 0; i < data2.m_ArrayLength; ++i)
	//{
	//	std::cout << data2.m_Col[i] << " ";
	//}
	//std::cout << std::endl;

	//for (uint32 i = 0; i < data2.m_Dimension; ++i)
	//{
	//	std::cout << data2.m_Fnz[i] << " ";
	//}
	//std::cout << std::endl;

	//std::cout << data2.m_ArrayLength << std::endl;
	//std::cout << data2.m_Dimension << std::endl;

	//double* b2 = new double[data2.m_Dimension];

	//std::ifstream file("../res/b.txt");
	//std::string str;
	//getline(file, str);
	//getline(file, str);
	//std::stringstream stream(str);
	//uint32 i = 0;
	//while (stream >> b2[i++]) {}

	//double* x2 = new double[data2.m_Dimension];
	//for (uint32 i = 0; i < data2.m_Dimension; ++i)
	//{
	//	x2[i] = 0.0;
	//}

	//S_BICGSTAB solverS_BICGSTAB(std::move(data2), x2, b2, dimension, 50000, 1e-6);
	//solverS_BICGSTAB.start();
	//std::cout << solverS_BICGSTAB.getTime() << std::endl;
	//std::cout << solverS_BICGSTAB.getIter() << std::endl;

	//P_BiCGSTAB solverP_BICGSTAB(std::move(data2), x2, b2, dimension, 50000, 1e-6);
	//solverP_BICGSTAB.start();

	//std::cout << solverP_BICGSTAB.getTime() << std::endl;
	//std::cout << solverP_BICGSTAB.getIter() << std::endl;

	//uint32 error = 0;
	//for (uint32 i = 0; i < 5000; ++i)
	//{
	//	if (parallel[i] != sequence[i])
	//	{
	//		std::cout << parallel[i] << "   " << sequence[i] << std::endl;
	//		++error;
	//	}
	//}
	//std::cout << "error: " << error << std::endl;

	//for (uint32 i = 0; i < dimension; ++i)
	//{
	//	std::cout << x2[i] << " ";
	//}
	//std::cout << std::endl;

	delete[] x;
	delete[] b;
	//delete[] x1;
	//delete[] b1;
	//delete[] x2;
	//delete[] b2;

	return 0;
}